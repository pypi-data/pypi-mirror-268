#include "hip/hip_runtime.h"
#include "ConvertToFloat.h"

#include "GLUtils.h"
#include <hip/hip_runtime.h>
#include "CudaImage.h"

using namespace std;

// convert "S10.5" flow vectors (int16s with the first 10 bits as integer part, last 5 bits as a fractional part)
// to floating point representation
__global__
void s105_vec_to_float_image(CudaImageRef<S105Vector> s105_vec_image, CudaImageRef<Point<float>> float_img) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < s105_vec_image.width && y < s105_vec_image.height) {
        float_img.write({x, y}, s105_to_point_float(s105_vec_image.read({x, y})));
    }
}

/*__global__
void uint8_to_float(CudaImageRef<uint8_t> uint8_img, CudaImageRef<float> float_img) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < uint8_img.width && y < uint8_img.height) {
        float_img.write({x, y}, static_cast<float>(uint8_img.read({x, y})));
    }
}*/

void ConvertToFloat::process_image(const std::shared_ptr<OpticalFlowImage> &img) {
    /*{
        auto uint8_img = img->GetGpuImageRef();
        img->float_img = std::make_shared<CudaImageOwned<float, 1>>(uint8_img.width, uint8_img.height);

        dim3 block_size(16, 16);
        dim3 grid_size((uint8_img.width + block_size.x - 1) / block_size.x, (uint8_img.height + block_size.y - 1) / block_size.y);
        uint8_to_float<<<grid_size, block_size>>>(uint8_img, img->float_img->ref());
    }*/

    auto s105_vec_image = img->GetGpuFlowVectorsRef();
    if (s105_vec_image) {
        img->float_vecs = std::make_shared<CudaImageOwned<Point<float>>>(s105_vec_image->width, s105_vec_image->height);

        dim3 block_size(16, 16);
        dim3 grid_size((s105_vec_image->width + block_size.x - 1) / block_size.x,
                       (s105_vec_image->height + block_size.y - 1) / block_size.y);
        s105_vec_to_float_image<<<grid_size, block_size>>>(*s105_vec_image, img->float_vecs->ref());
    }
}
