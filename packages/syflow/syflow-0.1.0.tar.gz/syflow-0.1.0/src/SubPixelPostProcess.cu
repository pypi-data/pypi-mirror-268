#include "hip/hip_runtime.h"
#include "SubPixelPostProcess.h"
#include "OpticalFlowUploader.h"

using namespace std;

struct DotWindows {
    uint32_t up{0};
    uint32_t down{0};
    uint32_t left{0};
    uint32_t center{0};
    uint32_t right{0};
};

__device__ DotWindows dot_windows(CudaImageRef<uint8_t> src_img, CudaImageRef<uint8_t> dst_img, PointU32 src_center, PointU32 dst_center, uint32_t window_size) {
    PointU32 top_left_offset{window_size / 2, window_size / 2};
    if (top_left_offset.x + 1 > src_center.x || top_left_offset.x + 1 > dst_center.x ||
        top_left_offset.y + 1 > src_center.y || top_left_offset.y + 1 > dst_center.y) {
        // would go negative outside of image bounds
        return {};
    }

    Point src_top_left = src_center - top_left_offset;
    Point dst_top_left = dst_center - top_left_offset;
    if (src_top_left.x + window_size + 1 > src_img.width || dst_top_left.x + window_size + 1 > dst_img.width ||
        src_top_left.y + window_size + 1 > src_img.height || dst_top_left.y + window_size + 1 > dst_img.height) {
        // would go greater than image bounds, invalid
        return {};
    }

    DotWindows sums;
    uint8_t* src_addr = src_img.get_addr(src_top_left);
    uint8_t* dst_addr = dst_img.get_addr(dst_top_left);
    for (uint32_t y_off = 0; y_off < window_size; y_off++) {
        uint32_t left = *(dst_addr - 1);
        uint32_t center = *dst_addr;
        uint32_t right = *(dst_addr + 1);
        for (uint32_t x_off = 0; x_off < window_size; x_off++) {
            uint32_t src_val = *src_addr;
            sums.up += src_val * *(dst_addr - dst_img.pitch);
            sums.down += src_val * *(dst_addr + dst_img.pitch);
            sums.left += src_val * left;
            sums.center += src_val * center;
            sums.right += src_val * right;

            src_addr += 1;
            dst_addr += 1;

            left = center;
            center = right;
            right = *(dst_addr + 1);
        }
        src_addr += src_img.pitch;
        dst_addr += dst_img.pitch;
    }

    return sums;
}

__global__
void refine(CudaImageRef<uint8_t> src_img,
            CudaImageRef<uint8_t> dst_img,
            CudaImageRef<S105Vector> vectors, uint32_t block_size,
            CudaImageRef<Point<float>> out_img, uint32_t window_size, uint32_t boundary_skip_size) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x + boundary_skip_size;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y + boundary_skip_size;

    if (x < vectors.width && y < vectors.height) {
        PointU32 src_pt{x * block_size + (block_size - 1) / 2, y * block_size + (block_size - 1) / 2};
        auto vec_initial_guess_s105 = vectors.read({x, y});
        auto vec_initial_guess_rounded = s105_to_point_rounded(vec_initial_guess_s105);

        auto dst_pt_int = PointI32{static_cast<int32_t>(src_pt.x), static_cast<int32_t>(src_pt.y)} + vec_initial_guess_rounded;
        if (dst_pt_int.x <= 0 || dst_pt_int.y <= 0 || dst_pt_int.x >= (dst_img.width - 1) || dst_pt_int.y >= (dst_img.height - 1)) {
            // vector (+/- 1px) goes outside of destination image, don't do refinement
            out_img.write({x, y}, s105_to_point_float(vec_initial_guess_s105));
            return;
        }
        PointU32 dst_pt{static_cast<uint32_t>(dst_pt_int.x), static_cast<uint32_t>(dst_pt_int.y)};

        auto sums = dot_windows(src_img, dst_img, src_pt, dst_pt, window_size);

        if (sums.up == 0 || sums.down == 0 || sums.left == 0 || sums.right == 0 || sums.center == 0) {
            // invalid dot product, don't do refinement
            out_img.write({x, y}, s105_to_point_float(vec_initial_guess_s105));
        } else {
            // fit [left, center, right] to a gaussian distribution
            float x_mean = (logf((float) sums.left) - logf((float) sums.right)) / (2 * logf((float) sums.left) - 4 * logf((float) sums.center) + 2 * logf((float) sums.right));
            float y_mean = (logf((float) sums.up) - logf((float) sums.down)) / (2 * logf((float) sums.up) - 4 * logf((float) sums.center) + 2 * logf((float) sums.down));

            Point<float> refined = {static_cast<float>(vec_initial_guess_rounded.x) + x_mean,
                                    static_cast<float>(vec_initial_guess_rounded.y) + y_mean};
            out_img.write({x, y}, refined);
        }
    }
}

// copy the top and bottom border, including corners
__global__ void copy_border_top_bottom(CudaImageRef<S105Vector> vectors, CudaImageRef<Point<float>> out_img, uint32_t boundary_skip_size) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= out_img.width || y >= 2 * boundary_skip_size) {
        return;
    }
    if (y >= boundary_skip_size) {
        y = y - 2*boundary_skip_size + out_img.height;
    }
    out_img.write({x, y}, s105_to_point_float(vectors.read({x, y})));
}

// copy the left and right borders, excluding the corners
__global__ void copy_border_sides(CudaImageRef<S105Vector> vectors, CudaImageRef<Point<float>> out_img, uint32_t boundary_skip_size) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= 2 * boundary_skip_size || y >= out_img.height - 2 * boundary_skip_size) {
        return;
    }
    if (x >= boundary_skip_size) {
        x = x - 2*boundary_skip_size + out_img.width;
    }
    out_img.write({x, y}, s105_to_point_float(vectors.read({x, y})));
}

void SubPixelPostProcess::process_image(const std::shared_ptr<OpticalFlowImage> &img) {
    auto orig_vecs = img->GetGpuFlowVectorsRef();
    if (orig_vecs) {
        auto out = std::make_shared<CudaImageOwned<Point<float>>>(orig_vecs->width, orig_vecs->height);

        uint32_t window_size = 32;
        // don't do postprocessing around a fixed-sized border, because the window can't fit in the source image
        uint32_t boundary_skip_size = window_size / img->block_size;
        // expand boundary even more to speed up (optional)
        boundary_skip_size *= 2;

        // copy the border
        dim3 block_size{16, 16};
        dim3 grid_size{(orig_vecs->width + block_size.x - 1) / block_size.x,
                       (2 * boundary_skip_size + block_size.y - 1) / block_size.y};
        // copy top and bottom border, including all 4 corners
        copy_border_top_bottom<<<grid_size, block_size>>>(*orig_vecs, out->ref(), boundary_skip_size);

        // copy the columns on the left and right side, but not the corners
        if (orig_vecs->height > 2 * boundary_skip_size) {
            grid_size = dim3{(2 * boundary_skip_size + block_size.x - 1) / block_size.x,
                             (orig_vecs->height - 2 * boundary_skip_size + block_size.y - 1) / block_size.y};
            copy_border_sides<<<grid_size, block_size>>>(*orig_vecs, out->ref(), boundary_skip_size);
        }

        if (2 * boundary_skip_size < orig_vecs->width && 2 * boundary_skip_size < orig_vecs->height) {
            // check that the image isn't so small that there is no region outside the boundary
            grid_size = dim3{(orig_vecs->width - 2 * boundary_skip_size + block_size.x - 1) / block_size.x,
                             (orig_vecs->height - 2 * boundary_skip_size + block_size.y - 1) / block_size.y};
            refine<<<grid_size, block_size>>>(this->last_image->GetGpuImageRef(), img->GetGpuImageRef(),
                                              *orig_vecs, img->block_size, out->ref(), window_size, boundary_skip_size);
        }
    }

    this->last_image = img;

    this->push_blocking(img);
}

void SubPixelPostProcess::on_thread_stop() {
    this->last_image.reset();
}
